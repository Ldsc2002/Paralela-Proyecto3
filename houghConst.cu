#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <vector>
#include <jpeglib.h>
#include <string>
#include "pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

// Constant memory for CUDA kernel
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

/* 
Function: CPU_HoughTran
Purpose: CPU implementation of Hough Transform
Arguments:
    pic - input image
    w - width of input image
    h - height of input image
    acc - output accumulator
Returns:
    None
 */
void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc) {
    // Calculate the maximum possible radius
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;  
    
    // Initialize the accumulator
    *acc = new int[rBins * degreeBins];            
    memset(*acc, 0, sizeof(int) * rBins * degreeBins); 
    
    // Calculate the center of the image
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    // For each pixel in the image
    for (int i = 0; i < w; i++) 
        for (int j = 0; j < h; j++) {
            int idx = j * w + i;
            
            if (pic[idx] > 0) {
                // Calculate the radius for each degree
                int xCoord = i - xCent;
                int yCoord = yCent - j;
                float theta = 0;       

                // For each degree  
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
                    // Calculate the radius
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;

                    // Increment the accumulator
                    (*acc)[rIdx * degreeBins + tIdx]++;
                    theta += radInc;
                }
            }
        }
}

/* 
Function: GPU_HoughTranConst
Purpose: GPU implementation of Hough Transform using constant memory
Arguments:
    pic - input image
    w - width of input image
    h - height of input image
    acc - output accumulator
    rMax - maximum possible radius
    rScale - scaling factor for radius
Returns:
    None
 */
__global__ void GPU_HoughTranConst(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return; // out of bounds

    // Calculate the center of the image
    int xCent = w / 2;
    int yCent = h / 2;
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        // For each degree
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;

            // Increment the accumulator
            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

/* 
Function: main
Purpose: Driver function for Hough Transform
Arguments:
    argc - number of command line arguments
    argv - command line arguments
Returns:
    None
 */
int main (int argc, char **argv) {
    // Initilize variables
    int i;
    std::string arg = argv[2];
    std::size_t pos;
    int threshold = std::stoi(arg,&pos);

    // Read input image
    PGMImage* inImg = new PGMImage(argv[1], 1);

    int *cpuht;

    // Get image dimensions
    int w = inImg->getXDim();
    int h = inImg->getYDim();

    hipEvent_t start, stop;

    // Create CUDA events for timing purposes
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory for constant memory
    hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
    hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

    // CPU calculation
    CPU_HoughTran(inImg->getPixels(), w, h, &cpuht);

    // Allocate memory for constant memory
    float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
    float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
    float rad = 0;

    // Calculate the cosine and sine values for each degree
    for (i = 0; i < degreeBins; i++) {
        pcCos[i] = cos (rad);
        pcSin[i] = sin (rad);
        rad += radInc;
    }

    // Calculate the maximum possible radius
    float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    // Copy constant memory to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

    // Setup and copy data from host to device
    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    // Get image data
    h_in = inImg->getPixels();

    h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

    // Allocate memory on device
    hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
    hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
    hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

    // Launch kernel
    int blockNum = ceil (w * h / 256);
    hipEventRecord(start);
    GPU_HoughTranConst <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);

    // Get results from device
    hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    
    // Check for errors
    for (i = 0; i < degreeBins * rBins; i++) {
        if (cpuht[i] != h_hough[i]) {
            printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
        }
    }

    printf("Done!\n");
    std::vector<std::pair<int, int>> lines;
    for (i = 0; i < degreeBins * rBins; i++){
        if (h_hough[i] > threshold) {
            int my_r = i / degreeBins;
            int my_th = i % degreeBins;
            std::pair<int, int> line = {my_r, my_th};
            lines.push_back(line);
        }
    }

    // Write output image
    inImg->write("ConstOutput.jpeg", lines, radInc, rBins);

    // Print timing results
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Milliseconds: %.3f ms\n" ,milliseconds);
    printf("Seconds: %d.%.3d s\n", (int)milliseconds/1000, (int)milliseconds%1000);

    // Free memory
    hipFree((void *) d_Cos);
    hipFree((void *) d_Sin);
    hipFree((void *) d_in);
    hipFree((void *) d_hough);
    delete[] pcCos;
    delete[] pcSin;
    delete inImg;
    hipDeviceReset();

  return 0;
}
